#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "Timer.hpp"

using namespace std;
using LOFAR::NSTimer;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(double* deviceA, double* deviceB, double* deviceResult) 
{
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
	
    deviceResult[index] = deviceA[index] + deviceB[index];
}


void vectorAddCuda(int n, double* a, double* b, double* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    double* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(double)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    double* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(double)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    double* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(double)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    NSTimer kernelTime = NSTimer("kernelTime", false, false);
    NSTimer memoryTime = NSTimer("memoryTime", false, false);

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(double), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime.start();
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipDeviceSynchronize();
    kernelTime.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(double), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    cout << fixed << setprecision(6);
    cout << "vector-add (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
    cout << "vector-add (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;
}


int main(int argc, char* argv[]) {
    int n = 8 * 1024 * 1024;
    NSTimer vectorAddTimer("vector add timer");
    double* a = new double[n];
    double* b = new double[n];
    double* result = new double[n];
    double* resultCpu = new double[n];

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = i;
    }

	for(int i=0; i<n; i++) {
		resultCpu[i] = a[i] + b[i] ;
	}

    vectorAddCuda(n, a, b, result);

    // verify the resuls
    for(int i=0; i<n; i++) {
        if( fabs(resultCpu[i] - result[i]) > 0.01  ) {
            cout	<< "error in results! Element " << i << " is " << result[i] 
					<< ", but should be " << resultCpu[i] << endl;
            exit(1);
        }
    }
    cout << "results OK!" << endl;
            
    delete[] a;
    delete[] b;
    delete[] result;
    
    return 0;
}
