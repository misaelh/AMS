#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "Timer.hpp"
using namespace std;
using LOFAR::NSTimer;

unsigned int SIZE;

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

void matrixMultiplyCPU()
{
	int i, j, k;
	float  **mat1 = new float * [SIZE];
	for(i = 0; i < SIZE; ++i)
		mat1[i] = new float[SIZE];	
	
	float  **mat2 = new float * [SIZE];
	for(i = 0; i < SIZE; ++i)
		mat2[i] = new float[SIZE];	

	float** prod= new float * [SIZE];
	for(i = 0; i < SIZE; ++i)
		prod[i] = new float[SIZE];	
	
	
	for (i = 0;i < SIZE; i++)
	{
		for (j = 0; j < SIZE; j++)
		{
			mat1[i][j] = 1.0;
		}
	}
	
	for(i = 0; i < SIZE; i++)
	{
		for (j = 0; j < SIZE; j++)
		{
			mat2[i][j] = 0.01;
		}
	}
	
	NSTimer cpuTime = NSTimer("cpuTime", false, false);
	cpuTime.start();
	for (i = 0;i < SIZE; i++)
	{
		for (j = 0; j < SIZE; j++)
		{
			prod[i][j]=0;
			for(k=0;k<SIZE;k++)
				prod[i][j] = prod[i][j]+mat1[i][k] * mat2[k][j];
		}
	}
	cpuTime.stop();
	cout << fixed << setprecision(6);
	cout << "total CPU time : \t\t" << cpuTime.getElapsed() << " seconds." << endl;	
	
/*	for(i = 0; i < SIZE; i++)
	{
		for (j = 0; j < SIZE; j++)
		{
			cout<<prod[i][j] << "	";
		}
		cout<<endl;
	}*/
	
}


/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiplyGPU(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer memoryTime = NSTimer("memoryTime", false, false);
	

    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);
	
	
    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	memoryTime.start();
    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	memoryTime.stop();
	
    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    
    memoryTime.start();
    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	memoryTime.stop();
	
    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16)
    {
        matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
        matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");

    // Execute the kernel
	kernelTime.start();
	if (block_size == 16)
	{
		matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}
	else
	{
		matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}
	hipDeviceSynchronize();	
    kernelTime.stop();

    // Copy result from device to host
    memoryTime.start();	
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
	memoryTime.stop();
	
    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: \n");
    bool correct = true;
    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        if (fabs(h_C[i] - (dimsA.x * valB)) > 1e-5)
        {
//             printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > 1e-5\n", i, h_C[i], dimsA.x*valB);
            correct = false;
        }
    }
    printf("%s\n", correct ? "OK" : "FAIL");
	
	cout << fixed << setprecision(6);
	cout << "kernel time : \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	cout << "memory time : \t\t" << memoryTime.getElapsed() << " seconds." << endl;	
	cout << "total GPU time : \t\t" << kernelTime.getElapsed() + memoryTime.getElapsed() << " seconds." << endl;

	
    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNote: For peak performance, please refer to the matrixMulCUBLAS example.\n");

    hipDeviceReset();

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");
	if(argc == 2)
		SIZE = atoi(argv[1]);
	else
		SIZE=256;

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;
    hipSetDevice(devID);

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;

    dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
    dim3 dimsB(5*4*block_size, 5*2*block_size, 1);

    // width of Matrix A
    dimsA.x = SIZE;

    // height of Matrix A
    dimsA.y = SIZE;

    // width of Matrix B
    dimsB.x = SIZE;

    // height of Matrix B
    dimsB.y = SIZE;

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiplyGPU(argc, argv, block_size, dimsA, dimsB);
	
	matrixMultiplyCPU();
	
    exit(matrix_result);
}
