#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

/* Utility function/macro, used to do error checking.
Use this function/macro like this:
checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
And to check the result of a kernel invocation:
checkCudaCall(hipGetLastError());
*/
#define checkCudaCall(result) {                                     \
	if (result != hipSuccess){                                     \
	cerr << "cuda error: " << hipGetErrorString(result);       \
	cerr << " in " << __FILE__ << " at line "<< __LINE__<<endl; \
	exit(1);                                                    \
	}                                                               \
}

__global__ void rgb2grayCudaKernel(unsigned char *deviceImage, unsigned char *deviceResult, const int height, const int width){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	int i = globalThreadNum;

	float grayPix = 0.0f;
	float r = static_cast< float >(deviceImage[i]);
	float g = static_cast< float >(deviceImage[(width * height) + i]);
	float b = static_cast< float >(deviceImage[(2 * width * height) + i]);
	grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

	deviceResult[i] = static_cast< unsigned char > (grayPix);
}

void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) {

	unsigned char *deviceImage;
	unsigned char *deviceResult;

	int initialBytes = width * height * 3;  
	int endBytes =  width * height * sizeof(unsigned char);

	hipMalloc((void**) &deviceImage, initialBytes);
	hipMalloc((void**) &deviceResult, endBytes);
	hipMemset(deviceResult, 0, endBytes);
	hipMemset(deviceImage, 0, initialBytes);

	hipError_t err = hipMemcpy(deviceImage, inputImage, initialBytes, hipMemcpyHostToDevice);    

	// Convert the input image to grayscale 
	rgb2grayCudaKernel<<<width * height / 256, 256>>>(deviceImage, deviceResult, height, width);
	hipDeviceSynchronize();

	hipMemcpy(grayImage, deviceResult, endBytes, hipMemcpyDeviceToHost);

	////// Sequential
	for ( int y = 0; y < height; y++ ) {
		for ( int x = 0; x < width; x++ ) {
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);
			grayImage[(y * width) + x] = static_cast< unsigned char > (grayPix);
		}
	}

	//compare sequential and cuda and print pixels that are wrong
	for (int i = 0; i < endBytes; i++)
	{
		if (grayImage[i] != grayImage[i])
			cout << i << "-" << static_cast< unsigned int >(grayImage[i]) <<
			" should be " << static_cast< unsigned int >(grayImage[i]) << endl;
	}

	hipFree(deviceImage);
	hipFree(deviceResult);
}

void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

			grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
		}
	}
	// /Kernel
	kernelTime.stop();

	cout << fixed << setprecision(6);
	cout << "rgb2gray (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/////////////////////////////////////
/*
__global__ void histogram1DCudaKernel
{
}
*/ 
__global__ void histogram1DCudaKernel(unsigned char *grayImg, unsigned int *hist, const int no_of_bins, const int width, const int height){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	int startIdxHist = globalThreadNum*no_of_bins;
	int startIdxImg = globalThreadNum*width;

	for(int i = startIdxImg; i < startIdxImg + width && i<width*height; i++) {
		hist[startIdxHist+grayImg[i]]++;
	}
}

__global__ void sumHistCuda(unsigned int *histArray, unsigned int *hist, const int no_of_bins, const int height, const int width){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	for(int i = 0; i < height; i++) {
		hist[globalThreadNum] += histArray[i*no_of_bins+globalThreadNum];
	}
}

void histogram1DCuda(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int BAR_WIDTH)
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	unsigned int *histArray;
	unsigned int hist[256] = {0};
	histArray = (unsigned int*)malloc(height*HISTOGRAM_SIZE*sizeof(unsigned int));
	memset(histArray, 0, height*HISTOGRAM_SIZE*sizeof(unsigned int));

	unsigned char *grayImgCuda;
	unsigned int *histArrayComputedCuda;
	unsigned int *histCuda;

	hipMalloc((void **) &histArrayComputedCuda, height*HISTOGRAM_SIZE*sizeof(unsigned int));
	hipMemset(histArrayComputedCuda, 0, height*HISTOGRAM_SIZE*sizeof(unsigned int));
	hipMalloc((void **) &grayImgCuda, width*height*sizeof(unsigned char));

	dim3 gridSize(16,1,1);
	dim3 blockSize(16,16,1);
	hipMemcpy(grayImgCuda,grayImage,sizeof(unsigned char)*height*width,hipMemcpyHostToDevice);
	histogram1DCudaKernel<<<gridSize, blockSize>>>(grayImgCuda, histArrayComputedCuda, HISTOGRAM_SIZE, width, height);
	hipError_t err = hipDeviceSynchronize();
	err = hipMemcpy(histArray, histArrayComputedCuda, height*HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
	err = hipMalloc((void **)&histCuda,HISTOGRAM_SIZE*sizeof(unsigned int));
	err = hipMemset(histCuda, 0, HISTOGRAM_SIZE*sizeof(unsigned int));

	dim3 gridSize2(16,1,1);
	dim3 blockSize2(16,1,1);
	//err = hipMemcpy(histArrayComputedCuda, histArray, height*HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyHostToDevice);
	sumHistCuda<<<gridSize, blockSize2>>>(histArrayComputedCuda, histCuda, 256, height, width);
	err = hipDeviceSynchronize();
	err = hipMemcpy(histogram, histCuda, HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	cout << fixed << setprecision(6);
	cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	hipFree(grayImgCuda);
	hipFree(histArrayComputedCuda);
	hipFree(histCuda);
}

void histogram1D(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int BAR_WIDTH) 
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
		}
	}
	// /Kernel
	kernelTime.stop();

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	cout << fixed << setprecision(6);
	cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/////////////////////////////////////
/*
__global__ void contrast1DKernel
{
}
*/

/*
void contrast1DCuda
{
}
*/

void contrast1D(unsigned char *grayImage, const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int CONTRAST_THRESHOLD) 
{
	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for (int x = 0; x < width; x++ ) 
		{
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min ) 
			{
				pixel = 0;
			}
			else if ( pixel > max ) 
			{
				pixel = 255;
			}
			else 
			{
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}

			grayImage[(y * width) + x] = pixel;
		}
	}
	// /Kernel
	kernelTime.stop();

	cout << fixed << setprecision(6);
	cout << "contrast1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/////////////////////////////////////

__global__ void triangularSmoothKernel(unsigned char *grayScale, unsigned char *smoothened, unsigned int width, unsigned int height, float *window)
{
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	int pixelPos = globalThreadNum;
	int modWidth = pixelPos%width;
	int modHeight = (pixelPos/width)%height;

	int x, y;
	int el_sum = 0;
	float smoothened_f = 0.0f;

	int x_start = 0, x_end = 5, y_start = 0, y_end = 5;

	if(pixelPos >= width * height)
		return;

	if(modWidth <=1)
		x_start = 2-modWidth;

	if(modWidth >= width - 2)
		x_end = 5 + modWidth - width;

	if(modHeight <=1)
		y_start = 2-modHeight;

	if(modHeight >= height - 2)
		y_end = 5 + modHeight - height;

	for(y = y_start; y < y_end; y++){
		for(x = x_start; x < x_end; x++) {
			smoothened_f += window[5*y+x] * grayScale[pixelPos+x-2+(y-2)*width];
			el_sum +=window[5*y+x];
		}
	}
	smoothened_f/=el_sum;
	smoothened[pixelPos] = smoothened_f;
}

void triangularSmoothCuda(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
	const float *filter)
{
	unsigned char *cudaImGray, *cudaEnhanced;
	float *cudaFilter;

	hipMalloc((void**)&cudaImGray, height*width*sizeof(unsigned char));
	hipMalloc((void**)&cudaEnhanced, height*width*sizeof(unsigned char));
	hipMalloc((void**)&cudaFilter, 25*sizeof(float));

	hipMemcpy(cudaImGray, grayImage, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemset(cudaEnhanced, 0, height*width*sizeof(unsigned char));
	hipMemcpy(cudaFilter, filter, 25*sizeof(float), hipMemcpyHostToDevice);

	dim3 gridSize2(65535, 1,1);
	dim3 blockSize2(16,31,1);

	triangularSmoothKernel<<<gridSize2, blockSize2>>> (cudaImGray, cudaEnhanced, width, height, cudaFilter);
	hipError_t err = hipMemcpy(smoothImage, cudaEnhanced ,height*width*sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	hipFree(cudaImGray);
	hipFree(cudaEnhanced);
	hipFree(cudaFilter);
}

void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
	const float *filter) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			unsigned int filterItem = 0;
			float filterSum = 0.0f;
			float smoothPix = 0.0f;

			for ( int fy = y - 2; fy < y + 3; fy++ ) 
			{
				for ( int fx = x - 2; fx < x + 3; fx++ ) 
				{
					if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) 
					{
						filterItem++;
						continue;
					}

					smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
					filterSum += filter[filterItem];
					filterItem++;
				}
			}

			smoothPix /= filterSum;
			smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
	}
	// /Kernel
	kernelTime.stop();

	cout << fixed << setprecision(6);
	cout << "triangularSmooth (cpu): \t" << kernelTime.getElapsed() << " seconds." << endl;
}


