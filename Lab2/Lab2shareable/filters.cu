#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

/* Utility function/macro, used to do error checking.
Use this function/macro like this:
checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
And to check the result of a kernel invocation:
checkCudaCall(hipGetLastError());
*/
#define checkCudaCall(result) {                                     \
	if (result != hipSuccess){                                     \
	cerr << "cuda error: " << hipGetErrorString(result);       \
	cerr << " in " << __FILE__ << " at line "<< __LINE__<<endl; \
	exit(1);                                                    \
	}                                                               \
}

__constant__ float filter_sum = 35.0f;

__constant__ float filterCuda[] = {	1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 
						1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 
						1.0f, 2.0f, 3.0f, 2.0f, 1.0f, 
						1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 
						1.0f, 1.0f, 1.0f, 1.0f, 1.0f
};
__global__ void rgb2grayCudaKernel(unsigned char *deviceImage, unsigned char *deviceResult, const int height, const int width){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	int i = globalThreadNum;

	int grayPix = 0;
	grayPix = (30*deviceImage[i] + 59 * deviceImage[(width * height) + i] + 11 * deviceImage[(2 * width * height) + i])/100;
	deviceResult[i] = grayPix;
}

void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) {

	unsigned char *deviceImage;
	unsigned char *deviceResult;
	unsigned int imageSize = width * height;

	int initialBytes = width * height * 3 *sizeof(unsigned char);
	int endBytes =  width * height * sizeof(unsigned char);
	unsigned int xGridDim = 0, yGridDim=1;

	hipError_t err = hipMalloc((void**) &deviceImage, initialBytes);
	err = hipMalloc((void**) &deviceResult, endBytes);
	err = hipMemset(deviceResult, 0, endBytes);
	err = hipMemset(deviceImage, 0, initialBytes);

	err = hipMemcpy(deviceImage, inputImage, initialBytes, hipMemcpyHostToDevice);
	if(imageSize >= 8192*8192)
		if(imageSize%(8192*8192-1)==0)
			yGridDim = imageSize / (8192*8192-1);
		else
			yGridDim = imageSize / (8192*8192-1)+ 1;

	if(imageSize%1024==0)
		xGridDim = width*height / (1024*yGridDim);
	else
		xGridDim = width*height / (1024*yGridDim) + 1;

	// Convert the input image to grayscale 
	dim3 grid(xGridDim,yGridDim,1);
	dim3 block(32,32,1);

	rgb2grayCudaKernel<<<grid, block>>>(deviceImage, deviceResult, height, width);
	err = hipDeviceSynchronize();

	err = hipMemcpy(grayImage, deviceResult, endBytes, hipMemcpyDeviceToHost);
	hipFree(deviceImage);
	hipFree(deviceResult);
}

void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

			grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
		}
	}
	// /Kernel
	kernelTime.stop();

	//cout << fixed << setprecision(6);
	//cout << "rgb2gray (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

__global__ void histogram1DCudaKernel(unsigned char *grayImg, unsigned int *hist, const int no_of_bins, const int width, const int height){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	int startIdxHist = globalThreadNum*no_of_bins;
	int startIdxImg = globalThreadNum*width;

	for(int i = startIdxImg; i < startIdxImg + width && i<width*height; i++) {
		hist[startIdxHist+grayImg[i]]++;
	}
}

__global__ void sumHistCuda(unsigned int *histArray, unsigned int *hist, const int no_of_bins, const int height, const int width){
	/* calculate the global thread id*/
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	for(int i = 0; i < height; i++) {
		hist[globalThreadNum] += histArray[i*no_of_bins+globalThreadNum];
	}
}

void histogram1DCuda(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int BAR_WIDTH)
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	unsigned int *histArray;
	unsigned int hist[256] = {0};
	histArray = (unsigned int*)malloc(height*HISTOGRAM_SIZE*sizeof(unsigned int));
	memset(histArray, 0, height*HISTOGRAM_SIZE*sizeof(unsigned int));

	unsigned char *grayImgCuda;
	unsigned int *histArrayComputedCuda;
	unsigned int *histCuda;
	unsigned int xGridDim = 0;

	hipMalloc((void **) &histArrayComputedCuda, height*HISTOGRAM_SIZE*sizeof(unsigned int));
	hipMemset(histArrayComputedCuda, 0, height*HISTOGRAM_SIZE*sizeof(unsigned int));
	hipMalloc((void **) &grayImgCuda, width*height*sizeof(unsigned char));

	if(height%1024==0)
		xGridDim = height / 1024;
	else
		xGridDim = height / 1024 + 1;

	dim3 gridSize(xGridDim,1,1);
	dim3 blockSize(32,32,1);
	hipMemcpy(grayImgCuda,grayImage,sizeof(unsigned char)*height*width,hipMemcpyHostToDevice);
	histogram1DCudaKernel<<<gridSize, blockSize>>>(grayImgCuda, histArrayComputedCuda, HISTOGRAM_SIZE, width, height);
	hipError_t err = hipDeviceSynchronize();
	err = hipMemcpy(histArray, histArrayComputedCuda, height*HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
	err = hipMalloc((void **)&histCuda,HISTOGRAM_SIZE*sizeof(unsigned int));
	err = hipMemset(histCuda, 0, HISTOGRAM_SIZE*sizeof(unsigned int));

	dim3 gridSize2(1,1,1);
	dim3 blockSize2(16,16,1);
	//err = hipMemcpy(histArrayComputedCuda, histArray, height*HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyHostToDevice);
	sumHistCuda<<<gridSize, blockSize2>>>(histArrayComputedCuda, histCuda, 256, height, width);
	err = hipDeviceSynchronize();
	err = hipMemcpy(histogram, histCuda, HISTOGRAM_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( unsigned int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	//cout << fixed << setprecision(6);
	//cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	hipFree(grayImgCuda);
	hipFree(histArrayComputedCuda);
	hipFree(histCuda);
}

void histogram1D(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int BAR_WIDTH) 
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
		}
	}
	// /Kernel
	kernelTime.stop();

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( unsigned int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	//cout << fixed << setprecision(6);
	//cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

__global__ void contrast1DCudaKernel(unsigned char *deviceImage, unsigned char *deviceResult, const int height, const int width,
								 unsigned int min, unsigned int max, float diff)
{
	int threadsPerBlock  = blockDim.x * blockDim.y;
    int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
    int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

    int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;
    int i = globalThreadNum;

unsigned int grayPix = static_cast< unsigned int >(deviceImage[i]);

	if ( grayPix < min ) 
	{
		grayPix = 0;
	}
	else if ( grayPix > max ) 
	{
		grayPix = 255;
	}
	else 
	{
		grayPix = (255 * (grayPix - min) / diff);
	}

	deviceResult[i] = static_cast< unsigned char > (grayPix);
}

void contrast1DCuda(unsigned char *grayImage, const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int CONTRAST_THRESHOLD) 
{
    unsigned char *deviceImage;
    unsigned char *deviceResult;
	unsigned int imageSize = width * height;

    int numBytes =  width * height * sizeof(unsigned char);
	unsigned int i = 0, xGridDim = 0, yGridDim = 1;
	unsigned int maxHist = 0;

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > maxHist ) 
		{
			maxHist = histogram[i];
		}
	}

	i=0;
	while ( (i < HISTOGRAM_SIZE) && ((histogram[i]*HISTOGRAM_SIZE)/maxHist < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	hipMalloc((void**) &deviceImage, numBytes);
    hipMalloc((void**) &deviceResult, numBytes);
    hipMemset(deviceResult, 0, numBytes);
    hipMemset(deviceImage, 0, numBytes);

    hipError_t err = hipMemcpy(deviceImage, grayImage, numBytes, hipMemcpyHostToDevice);    
	if(imageSize >= 8192*8192)
		if(imageSize%(8192*8192-1)==0)
			yGridDim = imageSize / (8192*8192-1);
		else
			yGridDim = imageSize / (8192*8192-1)+ 1;

	if(imageSize%1024==0)
		xGridDim = width*height / (1024*yGridDim);
	else
		xGridDim = width*height / (1024*yGridDim) + 1;

	// Convert the input image to grayscale 
	dim3 grid(xGridDim,yGridDim,1);
	dim3 block(32,32,1);
    // Convert the input image to grayscale 
    contrast1DCudaKernel<<<grid, block>>>(deviceImage, deviceResult, height, width, min, max, diff);
    hipDeviceSynchronize();

    hipMemcpy(grayImage, deviceResult, numBytes, hipMemcpyDeviceToHost);

}

void contrast1D(unsigned char *grayImage, const int width, const int height, 
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
	const unsigned int CONTRAST_THRESHOLD) 
{
	unsigned int i = 0;
	unsigned int maxHist = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	for ( unsigned int j = 0; j < HISTOGRAM_SIZE; j++ ) 
	{
		if ( histogram[j] > maxHist ) 
		{
			maxHist = histogram[j];
		}
	}

	while ( (i < HISTOGRAM_SIZE) && ((histogram[i]*HISTOGRAM_SIZE/maxHist) < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for (int x = 0; x < width; x++ ) 
		{
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min ) 
			{
				pixel = 0;
			}
			else if ( pixel > max ) 
			{
				pixel = 255;
			}
			else 
			{
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}

			grayImage[(y * width) + x] = pixel;
		}
	}
	// /Kernel
	kernelTime.stop();

	//cout << fixed << setprecision(6);
	//cout << "contrast1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

__global__ void triangularSmoothKernel(unsigned char *grayScale, unsigned char *smoothened, unsigned int width, unsigned int height)
{
	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	int pixelPos = globalThreadNum;
	int modWidth = pixelPos%width;
	int modHeight = (pixelPos/width);

	int x, y;
	float smoothened_0 = 0, smoothened_1 = 0, smoothened_2 = 0, smoothened_3 = 0, smoothened_4 = 0, smoothened_f = 0;
	bool unrollFlag = true;
	int width2pos = 2*width, width2minus = -width2pos;

	int x_start = 0, x_end = 5, y_start = 0, y_end = 5;

	if(pixelPos >= width * height)
		return;

	if((modWidth < 2) || (modWidth > width - 3)) {
		unrollFlag = false;
		if(modWidth < 2)
			x_start = 2 - modWidth;
		else
			x_end = 2 + width - modWidth;
	}

	if((modHeight < 2) || (modHeight > height - 3)) {
		unrollFlag = false;
		if(modHeight < 2)
			y_start = 2 - modHeight;
		else
			y_end = 2 + height - modHeight;
	}

	if(!unrollFlag){
		float el_sum = 0;
		for(y = y_start; y < y_end; y++){
			for(x = x_start; x < x_end; x++) {
				smoothened_f += filterCuda[5*y+x] * grayScale[pixelPos+x-2+(y-2)*width];
				el_sum += filterCuda[5*y+x];
			}
		}
		smoothened_f/=el_sum;
		smoothened[pixelPos] = smoothened_f;
	}
	else {
		smoothened_0 += filterCuda[0] * grayScale[pixelPos-2+width2minus];
		smoothened_1 += filterCuda[1] * grayScale[pixelPos-1+width2minus];
		smoothened_2 += filterCuda[2] * grayScale[pixelPos+0+width2minus];
		smoothened_3 += filterCuda[3] * grayScale[pixelPos+1+width2minus];
		smoothened_4 += filterCuda[4] * grayScale[pixelPos+2+width2minus];

		smoothened_0 += filterCuda[5] * grayScale[pixelPos-2-width];
		smoothened_1 += filterCuda[6] * grayScale[pixelPos-1-width];
		smoothened_2 += filterCuda[7] * grayScale[pixelPos+0-width];
		smoothened_3 += filterCuda[8] * grayScale[pixelPos+1-width];
		smoothened_4 += filterCuda[9] * grayScale[pixelPos+2-width];

		smoothened_0 += filterCuda[10] * grayScale[pixelPos-2];
		smoothened_1 += filterCuda[11] * grayScale[pixelPos-1];
		smoothened_2 += filterCuda[12] * grayScale[pixelPos+0];
		smoothened_3 += filterCuda[13] * grayScale[pixelPos+1];
		smoothened_4 += filterCuda[14] * grayScale[pixelPos+2];

		smoothened_0 += filterCuda[15] * grayScale[pixelPos-2+width];
		smoothened_1 += filterCuda[16] * grayScale[pixelPos-1+width];
		smoothened_2 += filterCuda[17] * grayScale[pixelPos+0+width];
		smoothened_3 += filterCuda[18] * grayScale[pixelPos+1+width];
		smoothened_4 += filterCuda[19] * grayScale[pixelPos+2+width];

		smoothened_0 += filterCuda[20] * grayScale[pixelPos-2+width2pos];
		smoothened_1 += filterCuda[21] * grayScale[pixelPos-1+width2pos];
		smoothened_2 += filterCuda[22] * grayScale[pixelPos+0+width2pos];
		smoothened_3 += filterCuda[23] * grayScale[pixelPos+1+width2pos];
		smoothened_4 += filterCuda[24] * grayScale[pixelPos+2+width2pos];
		
		smoothened_0 = smoothened_0 + smoothened_1;
		smoothened_3 = smoothened_3 + smoothened_4;
		smoothened_0 += smoothened_2 + smoothened_3;

		smoothened_0/=filter_sum;
		smoothened[pixelPos] = smoothened_0;
	}
}

void triangularSmoothCuda(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height)
{
	unsigned char *cudaImGray, *cudaEnhanced;
	unsigned int xGridDim = 0, yGridDim = 1;
	unsigned int imageSize = width * height;

	hipMalloc((void**)&cudaImGray, height*width*sizeof(unsigned char));
	hipMalloc((void**)&cudaEnhanced, height*width*sizeof(unsigned char));

	hipMemcpy(cudaImGray, grayImage, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemset(cudaEnhanced, 0, height*width*sizeof(unsigned char));

	if(imageSize >= 8192*8192)
		if(imageSize%(8192*8192-1)==0)
			yGridDim = imageSize / (8192*8192-1);
		else
			yGridDim = imageSize / (8192*8192-1)+ 1;

	if(imageSize%1024==0)
		xGridDim = width*height / (1024*yGridDim);
	else
		xGridDim = width*height / (1024*yGridDim) + 1;

	// Convert the input image to grayscale 
	dim3 grid(xGridDim,yGridDim,1);
	dim3 block(32,32,1);

	triangularSmoothKernel<<<grid, block>>> (cudaImGray, cudaEnhanced, width, height);
	hipError_t err = hipMemcpy(smoothImage, cudaEnhanced ,height*width*sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	hipFree(cudaImGray);
	hipFree(cudaEnhanced);
}

void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
	const float *filter) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			unsigned int filterItem = 0;
			float filterSum = 0.0f;
			float smoothPix = 0.0f;

			for ( int fy = y - 2; fy < y + 3; fy++ ) 
			{
				for ( int fx = x - 2; fx < x + 3; fx++ ) 
				{
					if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) 
					{
						filterItem++;
						continue;
					}

					smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
					filterSum += filter[filterItem];
					filterItem++;
				}
			}

			smoothPix /= filterSum;
			smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
	}
	// /Kernel
	kernelTime.stop();

	//cout << fixed << setprecision(6);
	//cout << "triangularSmooth (cpu): \t" << kernelTime.getElapsed() << " seconds." << endl;
}